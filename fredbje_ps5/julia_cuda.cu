
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>

#define XSIZE 2560
#define YSIZE 2048
#define MAXITER 255
#define PIXEL(i,j) ((i)+(j)*XSIZE)

typedef struct {
    double re;
    double im;
} complex_t;

typedef unsigned char uchar;

double walltime(){
	static struct timeval t;
	gettimeofday(&t, NULL);
	return (t.tv_sec + 1e-6 * t.tv_usec);
}

/* save 24-bits bmp file, buffer must be in bmp format: upside-down */
void savebmp(const char *name, uchar *buffer, int x, int y) {
	FILE *f = fopen(name, "w");
	if(!f) {
		printf("Error writing image to disk.\n");
		return;
	}
	unsigned int size = x*y*3 + 54;
	uchar header[54]={'B','M',size&255,(size>>8)&255,(size>>16)&255,size>>24,0,
                    0,0,0,54,0,0,0,40,0,0,0,x&255,x>>8,0,0,y&255,y>>8,0,0,1,0,24,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
	fwrite(header, 1, 54, f);
	fwrite(buffer, 1, XSIZE*YSIZE*3, f);
	fclose(f);
}

/* given iteration number, set a colour */
void fancycolour(uchar *p, int iter) {
	if(iter == MAXITER);
	else if(iter < 8) { p[0] = 128 + iter*16; p[1] = p[2] = 0; }
	else if(iter < 24) { p[0] = 255; p[1] = p[2] = (iter - 8)*16; }
	else if(iter < 160) { p[0] = p[1] = 255 - (iter - 24)*2; p[2] = 255; }
	else { p[0] = p[1] = (iter - 160)*2; p[2] = 255 - (iter - 160)*2; }
}

__device__ void square_complex(complex_t a, complex_t *res){
    (*res).re = a.re*a.re - a.im*a.im;
    (*res).im = 2*a.re*a.im;
}

__device__ void add_complex(complex_t a, complex_t b, complex_t *res){
    (*res).re = a.re + b.re;
    (*res).im = a.im + b.im;
}

__device__ void add_real(complex_t a, int b, complex_t *res){
    (*res).re = a.re + b;
	(*res).im = a.im;
}

__global__ void iterate_pixel(int *pixel, complex_t julia_C){

	// Calculate the range in the y-axis such that we preserve the aspect ratio
	double x_start = -2.01;
	double x_end = 1.0;
	double y_center = 1e-6;
	double step = (x_end - x_start)/XSIZE;
	double y_start = y_center - (step*YSIZE)/2;

	int x =  blockIdx.x * blockDim.x + threadIdx.x;
	int y =  blockIdx.y * blockDim.y + threadIdx.y;

	complex_t z;
	z.re = (x_start + step*x);
	z.im = (y_start + step*y);

	int iter = 0;
	complex_t z_squared;
	while(z.re*z.re + z.im*z.im < 4) {
		if(++iter==MAXITER){
			break;
		} else {
			square_complex(z, &z_squared);
			add_complex(z_squared, julia_C, &z);
		}
	}
	pixel[PIXEL(x,y)]=iter;
}

int main(int argc,char **argv) {
	if(argc==1) {
		puts("Usage: JULIA\n");
		puts("Input real and imaginary part. ex: ./julia 0.0 -0.8");
		return 0;
	}

	complex_t julia_C;
    julia_C.re = strtod(argv[1], NULL);
    julia_C.im = strtod(argv[2], NULL);


/*  // Runs much slower 513ms as opposed to 0.088ms without mallocManaged
	double start_gpu = walltime();

	int *pixeldouble start_gpu = walltime();;
	cudaMallocManaged(&pixel, XSIZE*YSIZE*sizeof(int));
	dim3 grid(XSIZE, YSIZE);
	iterate_pixel<<<grid, 1>>>(pixel, julia_C);
	cudaDeviceSynchronize();

	double end_gpu = walltime();
	printf("Computation complete. It took %7.3f ms\n", end_gpu - start_gpu);

  	//create nice image from iteration counts. take care to create it upside down (bmp format)
 	uchar *buffer = (uchar*)calloc(XSIZE*YSIZE*3, sizeof(uchar));
    for(int i = 0; i < XSIZE; i++) {
        for(int j = 0; j < YSIZE; j++) {
            int p = ((YSIZE - j - 1)*XSIZE + i)*3;
			fancycolour(buffer + p, pixel[PIXEL(i, j)]);
        }
    }
    savebmp("julia_cuda.bmp", buffer, XSIZE, YSIZE);

	cudaFree(pixel);
	free(buffer);

*/



	double start_gpu = walltime();

	int *device_pixel;
	int *host_pixel;
	host_pixel = (int*)malloc(XSIZE*YSIZE*sizeof(int));
	hipMalloc((void**)&device_pixel, XSIZE*YSIZE*sizeof(int));
	int num_threads = 32;
	dim3 grid_dim(XSIZE/num_threads, YSIZE/num_threads);
	dim3 block_dim(num_threads, num_threads);
	iterate_pixel<<<grid_dim, block_dim>>>(host_pixel, julia_C);
	hipMemcpy(host_pixel, device_pixel, XSIZE*YSIZE*sizeof(int), hipMemcpyDeviceToHost);

	double end_gpu = walltime();
	printf("Computation complete. It took %7.3f ms\n", end_gpu - start_gpu);

  	// create nice image from iteration counts. take care to create it upside down (bmp format)
 	uchar *buffer = (uchar*)calloc(XSIZE*YSIZE*3, sizeof(uchar));
    for(int i = 0; i < XSIZE; i++) {
        for(int j = 0; j < YSIZE; j++) {
            int p = ((YSIZE - j - 1)*XSIZE + i)*3;
			fancycolour(buffer + p, host_pixel[PIXEL(i, j)]);
        }
    }
    savebmp("julia_cuda.bmp", buffer, XSIZE, YSIZE);

	hipFree(device_pixel);
	free(host_pixel);
	free(buffer);

    return 0;
}
